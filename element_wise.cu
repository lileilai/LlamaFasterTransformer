#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

#define THREAD_PER_BLOCK 256

// transfer vector
#define FETCH_FLOAT2(pointer) (reinterpret_cast<float2*>(&(pointer))[0])
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

__global__ void add(float* a, float* b, float* c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    c[idx] = a[idx] + b[idx];
}
__global__ void vec2_add(float* a, float* b, float* c)
{
    int idx = (threadIdx.x + blockIdx.x * blockDim.x)*2;
    //c[idx] = a[idx] + b[idx];
    float2 reg_a = FETCH_FLOAT2(a[idx]);
    float2 reg_b = FETCH_FLOAT2(b[idx]);
    float2 reg_c;
    reg_c.x = reg_a.x + reg_b.x;
    reg_c.y = reg_a.y + reg_b.y;
    FETCH_FLOAT2(c[idx]) = reg_c;
}

__global__ void vec4_add(float* a, float* b, float* c)
{
    int idx = (threadIdx.x + blockIdx.x * blockDim.x)*4;
    //c[idx] = a[idx] + b[idx];
    float4 reg_a = FETCH_FLOAT4(a[idx]);
    float4 reg_b = FETCH_FLOAT4(b[idx]);
    float4 reg_c;
    reg_c.x = reg_a.x * reg_b.x;
    reg_c.y = reg_a.y * reg_b.y;
    reg_c.z = reg_a.z * reg_b.z;
    reg_c.w = reg_a.w * reg_b.w;
    FETCH_FLOAT4(c[idx]) = reg_c;
}

bool check(float *out,float *res,int n){
    for(int i=0;i<n;i++){
        if(out[i]!=res[i])
            return false;
    }
    return true;
}

int main(){
    const int N=32;
    float *a=(float *)malloc(N*sizeof(float));
    float *b=(float *)malloc(N*sizeof(float));
    float *out=(float *)malloc(N*sizeof(float));

    float *d_a;
    float *d_b;
    float *d_out;
    hipMalloc((void **)&d_a,N*sizeof(float));
    hipMalloc((void **)&d_b,N*sizeof(float));
    hipMalloc((void **)&d_out,N*sizeof(float));


    hipMemset(static_cast<void*>(d_out), 1, sizeof(float)*N); 

    float *res=(float *)malloc(N*sizeof(float));

    // for(int i=0;i<N;i++){
    //     a[i]=1;
    //     b[i]=i;
    //     res[i]=a[i] * b[i];
    // }

    // cudaMemcpy(d_a,a,N*sizeof(float),cudaMemcpyHostToDevice);
    // cudaMemcpy(d_b,b,N*sizeof(float),cudaMemcpyHostToDevice);

    // dim3 Grid( N/THREAD_PER_BLOCK/4, 1);
    // dim3 Block( THREAD_PER_BLOCK, 1);

    // int iter = 10;
    // for(int i=0; i<iter; i++){
    //     vec4_add<<<Grid,Block>>>(d_a, d_b, d_out);
    // }


    printf("before: \n");

    for(int i=0;i<10;i++)
        printf("%lf\t", out[i]);

    hipMemcpy(out,d_out,N*sizeof(float),hipMemcpyDeviceToHost);


    printf("\nafter: \n");

    for(int i=0;i<10;i++)
        printf("%lf\t", out[i]);



    // if(check(out,res,N))printf("the ans is right\n");
    // else{
    //     printf("the ans is wrong\n");
    //     for(int i=0;i<N;i++){
    //         printf("%lf ",out[i]);
    //     }
    //     printf("\n");
    // }

    hipFree(d_a);
    hipFree(d_out);
}